#include "hip/hip_runtime.h"
#include "ppm.h"
#include <math.h>


// Constant memory for convolution filter
__constant__ Filter filter_c;

/*
// the black and white kernel, each thread changes a pixel
__global__ void blackAndWhite(PPMPixel *imageData, PPMPixel *outputData, int width, int height) {

	int tx = threadIdx.x;
	int ty = blockIdx.x;


	if(ty < height && tx < width) {
		int i = ty*width + tx;
		int avg = (imageData[i].red + imageData[i].green + imageData[i].blue) / 3;

		outputData[i].red = avg;
		outputData[i].green = avg;
		outputData[i].blue = avg;
	}
}
*/

// determines mandelbrot status for each pixel between b1, b2
__global__ void mandelbrot(float b1x, float b1y, float b2x, float b2y, PPMPixel *outputData, int width, int height) {

	int tx = threadIdx.x;
	int ty = blockIdx.x;

	int x_res = blockDim.x; // x resolution
	int y_res = gridDim.x;	// y resolution

	if (ty < height && tx < width) {
		// determine complex coords of pixel in relation to b1, b2	
		int i = ty*width + tx;
		float x0 = (tx * (b2x-b1x)/x_res) + b1x; // Scaled to lie in Mandelbrot X scale (b1x, b2x)
		float y0 = (ty * (b2y-b1y)/y_res) + b1y; // Scaled to lie in Mandelbrot Y scale (b1y, b2y)
	
		// determine mandelbrot status of pixel using optimized method
		float x1 = 0;
		float y1 = 0;
		float x2 = 0;
		float y2 = 0;
		int iteration = 0;
		int max_iteration = 20;
	
		// z(n+1) = z(n)^2 + c
		while (x2 + y2 <= 4 && iteration < max_iteration) {
			y1 = 2*x1*y1 + y0; 
			x1 = x2 - y2 + x0; 
	
			x2 = x1*x1;
			y2 = y1*y1;
			iteration ++; 
		}
		
		// color according to iteration
		outputData[i].red = iteration;
		outputData[i].green = iteration;
		outputData[i].blue = iteration;
	}
}

/*
// the convolution kernel, each thread convolves for a pixel
__global__ void convolution(PPMPixel *imageData, PPMPixel *outputData, int width, int height)
{
    __shared__ PPMPixel imageData_s[INPUT_TILE_SIZE][INPUT_TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get output tile row and col
    int row_o = blockIdx.y * OUTPUT_TILE_SIZE + ty;
    int col_o = blockIdx.x * OUTPUT_TILE_SIZE + tx;

    // get input tile row and col
    int row_i = row_o - FILTER_SIZE / 2;
    int col_i = col_o - FILTER_SIZE / 2;

    // if input is in bounds read from global to shared memory
    if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width))
    {
        imageData_s[ty][tx] = imageData[row_i * width + col_i];
    }
    else // set pixel to black (all zero)
    {
        imageData_s[ty][tx].red = 0;
        imageData_s[ty][tx].blue = 0;
        imageData_s[ty][tx].green = 0;
    }

    __syncthreads();

    int red = 0, blue = 0, green = 0;

    // if in bounds calculate convolution for this pixel
    if ((ty < OUTPUT_TILE_SIZE) && (tx < OUTPUT_TILE_SIZE))
    {
        int i, j;
        for (i = 0; i < FILTER_SIZE; i++)
        {
            for (j = 0; j < FILTER_SIZE; j++)
            {
                red   += filter_c.data[j * FILTER_SIZE + i] * imageData_s[j + ty][i + tx].red;
                blue  += filter_c.data[j * FILTER_SIZE + i] * imageData_s[j + ty][i + tx].blue;
                green += filter_c.data[j * FILTER_SIZE + i] * imageData_s[j + ty][i + tx].green;
            }
        }

        // write value to output, saturate between 0 and 255
        if ((row_o < height) && (col_o < width))
        {
            outputData[row_o * width + col_o].red   = min( max( (int)(filter_c.factor * red   + filter_c.bias), 0), 255);
            outputData[row_o * width + col_o].blue  = min( max( (int)(filter_c.factor * blue  + filter_c.bias), 0), 255);
            outputData[row_o * width + col_o].green = min( max( (int)(filter_c.factor * green + filter_c.bias), 0), 255);
        }
    }
}
*/
